#include"hip/hip_runtime.h"
#include<cstdio>
#include<fstream>
#include<random>
using namespace std;

__host__ void init(float* mat, int row, int col) {
    // 初始化随机数生成器
    random_device rd;
    default_random_engine eng(rd());
    uniform_int_distribution<int> distr(0, 255);

    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            mat[i * col + j] = distr(eng);
        }
    }
}

// 保存卷积后的矩阵
__host__ void write_result(float* r, float* g, float *b, int row, int col, string filename){
    ofstream f(filename);
    if(f.is_open()){
        f << "red:\n";
        for(int i = 0; i < row; ++i){
            for(int j = 0; j < col; ++j){
                if(r[i * col + j] > 255){
                    r[i * col + j] = 255;
                }
                if(r[i * col + j] < 0){
                    r[i * col + j] = 0;
                }
                f << r[i * col + j]<<" ";
            }
            f << "\n";
        }
        f << "\n" << "green:\n";
        for(int i = 0; i < row; ++i){
            for(int j = 0; j < col; ++j){
                if(g[i * col + j] > 255){
                    g[i * col + j] = 255;
                }
                if(g[i * col + j] < 0){
                    g[i * col + j] = 0;
                }                
                f << g[i * col + j]<<" ";
            }
            f << "\n";
        }
        f << "\n" << "blue:\n";
        for(int i = 0; i < row; ++i){
            for(int j = 0; j < col; ++j){
                if(b[i * col + j] > 255){
                    b[i * col + j] = 255;
                }
                if(b[i * col + j] < 0){
                    b[i * col + j] = 0;
                }                
                f << b[i * col + j]<<" ";
            }
            f << "\n";
        }
    }
}

// image to column
__global__ void im2col(float* input, float* output, int f_w, int f_h, int w_in, int h_in, int w_out, int h_out, int s){
    int y = blockIdx.y * blockDim.y + threadIdx.y, x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x >= 0 && x < w_out && y >= 0 && y < h_out){
        int col_index = y * w_out + x;
        for(int k = 0; k < f_h; ++k){
            for(int p = 0; p < f_w; ++p){
                int row_index = k * f_w + p;
                int input_row = y * s + k, input_col = x * s + p;
                output[row_index * (w_out * h_out) + col_index] = input[input_row * w_in + input_col];
            }
         }
    }
}

// 卷积
__global__ void convolution_shared(float* input, float* output, float* filter, int w_out, int h_out, int f_w, int f_h){
    extern __shared__ float smem[];
    int y = blockIdx.y * blockDim.y + threadIdx.y, x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x >= 0 && x < w_out && y >= 0 && y < h_out){
        int index_1 = y * w_out + x;
        for(int i = 0; i <f_h; ++i){
            for(int j = 0; j < f_w; ++j){
                smem[i * f_w + j] = input[(i * f_w + j) * (w_out * h_out) + index_1];
            }
        }
        __syncthreads();
        float sum = 0;
        for(int i = 0; i < f_h; ++i){
            for(int j = 0; j < f_w; ++j){
                int index_2 = i * f_w + j;
                sum += smem[index_2] * filter[index_2];
            }
        }
        atomicAdd(&output[index_1], sum);
    }
}

int main(){
    int w, h, w_pad, h_pad, w_out, h_out, f_w = 3, f_h = 3, s, threads, blockDim_x = 1, blockDim_y = 1;
    printf("请输入图像大小：\n");
    scanf("%d %d", &h, &w);
    
    printf("请输入步长：\n");
    scanf("%d", &s);
    
    printf("请输入输出图像大小：\n");
    scanf("%d %d", &h_out, &w_out);
    
    printf("请输入每个线程块内的线程数：\n");
    scanf("%d", &threads);
    
    printf("请输入线程块的维度：\n");
    while (scanf("%d %d", &blockDim_y, &blockDim_x) == 2 && blockDim_y * blockDim_x != threads) {
        printf("输入的维度不符合要求，两个维度的乘积要等于每个块内的线程数，请重新输入：\n");
    }
    float *r, *g, *b, *r_pad, *g_pad, *b_pad, *r_res, *g_res, *b_res, *f_1, *f_2, *f_3, *r_col, *g_col, *b_col;
    hipHostMalloc(&r, sizeof(float) * w * h, hipHostMallocDefault);
    hipHostMalloc(&g, sizeof(float) * w * h, hipHostMallocDefault);
    hipHostMalloc(&b, sizeof(float) * w * h, hipHostMallocDefault);
    
    // 获取原始的图像三通道的像素矩阵
    init(r, h, w);
    init(g, h, w);
    init(b, h, w);
    
    int delta_w = (w_out - 1) * s - w + f_w, delta_h = (h_out - 1) * s - h + f_h;//根据输出维度、步长和卷积核大小，计算出需要额外填充的列数和行数
    bool flag = (delta_w != 0) && (delta_h != 0); //行和列是否都需要填充(由于宽高相等，因此行和列要么都要进行填充，要么都不需要进行填充)
    if(flag){
        // 计算上下左右外围需要填充的行数和列数，如果能被2整除，上下/左右各填充一半，如果不行则下方/右侧多填充一行/列
        int pad_half_w_1 = delta_w / 2, pad_half_w_2 = (delta_w % 2 == 0) ? delta_w / 2 : delta_w / 2 + 1;
        int pad_half_h_1 = delta_h / 2, pad_half_h_2 = (delta_h % 2 == 0) ? delta_h / 2 : delta_h / 2 + 1;
        w_pad = w + delta_w;
        h_pad = h + delta_h;
        hipHostMalloc(&r_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        hipHostMalloc(&g_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        hipHostMalloc(&b_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        for(int i = 0; i < h_pad; ++i){
            for(int j = 0; j < w_pad; ++j){
                int index = i * w_pad + j;
                if(i >= pad_half_h_1 && i < h_pad - pad_half_h_2 &&
                   j >= pad_half_w_1 && j < w_pad - pad_half_w_2){
                    int original_index = (i - pad_half_h_1) * w + (j - pad_half_w_1);
                    r_pad[index] = r[original_index];
                    g_pad[index] = g[original_index];
                    b_pad[index] = b[original_index];
                }
                else{
                    r_pad[index] = 0;
                    g_pad[index] = 0;
                    b_pad[index] = 0;
                }
            }
        }
        
    }
    else{//不用进行零填充,直接将前面的r、g、b复制给r_pad、g_pad、b_pad即可
        w_pad = w;
        h_pad = h;
        hipHostMalloc(&r_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        hipHostMalloc(&g_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        hipHostMalloc(&b_pad, sizeof(float) * w_pad * h_pad, hipHostMallocDefault);
        hipMemcpy(r_pad, r, w_pad * h_pad * sizeof(float), hipMemcpyHostToHost);
        hipMemcpy(g_pad, g, w_pad * h_pad * sizeof(float), hipMemcpyHostToHost);
        hipMemcpy(b_pad, b, w_pad * h_pad * sizeof(float), hipMemcpyHostToHost);
    }
    
    // 初始化3个三通道卷积核
    hipHostMalloc(&f_1, sizeof(float) * f_h * f_w * 3, hipHostMallocDefault);
    hipHostMalloc(&f_2, sizeof(float) * f_h * f_w * 3, hipHostMallocDefault);
    hipHostMalloc(&f_3, sizeof(float) * f_h * f_w * 3, hipHostMallocDefault);
    for(int i = 0; i < f_h * f_w * 3; ++i){
        f_1[i] = 0.01;
        f_2[i] = 0.05;
        f_3[i] = 0.09;
    }
    
    dim3 blockDim(blockDim_x, blockDim_y, 1);
    dim3 gridDim((int)((w_out + blockDim.x - 1)/blockDim.x), (int)((h_out + blockDim.y - 1)/blockDim.y));
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipHostMalloc(&r_res, sizeof(float) * h_out * w_out, hipHostMallocDefault);
    hipHostMalloc(&g_res, sizeof(float) * h_out * w_out, hipHostMallocDefault);
    hipHostMalloc(&b_res, sizeof(float) * h_out * w_out, hipHostMallocDefault);
    hipHostMalloc(&r_col, sizeof(float) * h_out * w_out * f_w * f_h, hipHostMallocDefault);
    hipHostMalloc(&g_col, sizeof(float) * h_out * w_out * f_w * f_h, hipHostMallocDefault);
    hipHostMalloc(&b_col, sizeof(float) * h_out * w_out * f_w * f_h, hipHostMallocDefault);
    hipEventRecord(start, 0);
    im2col<<<gridDim, blockDim>>>(r_pad, r_col, f_w, f_h, w_pad, h_pad, w_out, h_out, s);
    im2col<<<gridDim, blockDim>>>(g_pad, g_col, f_w, f_h, w_pad, h_pad, w_out, h_out, s);
    im2col<<<gridDim, blockDim>>>(b_pad, b_col, f_w, f_h, w_pad, h_pad, w_out, h_out, s);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(r_col, r_res, f_1, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(g_col, g_res, f_1+9, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(b_col, b_res, f_1+18, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(r_col, r_res, f_2, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(g_col, g_res, f_2+9, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(b_col, b_res, f_2+18, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(r_col, r_res, f_3, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(g_col, g_res, f_3+9, w_out, h_out, f_w, f_h);
    convolution_shared<<<gridDim, blockDim, sizeof(float) * f_w * f_h>>>(b_col, b_res, f_3+18, w_out, h_out, f_w, f_h);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    float ms = 0;
    hipEventElapsedTime(&ms, start, end);
    printf("图像大小:%d*%d;线程块维度:(%d,%d);访存方式:共享内存;所用时间:%f ms\n",h,w,blockDim_y,blockDim_x,ms);
    
    string output_filename = to_string(h) + "*" + to_string(w) + " " + "(" + to_string(blockDim_y) + "," +  to_string(blockDim_x) + ") im2col shared.txt";
    write_result(r_res, g_res, b_res, h_out, w_out, output_filename);
    hipHostFree(r);
    hipHostFree(g);
    hipHostFree(b);
    hipHostFree(r_res);
    hipHostFree(g_res);
    hipHostFree(b_res);
    hipHostFree(r_pad);
    hipHostFree(g_pad);
    hipHostFree(b_pad);
    hipHostFree(r_col);
    hipHostFree(g_col);
    hipHostFree(b_col);
    hipHostFree(f_1);
    hipHostFree(f_2);
    hipHostFree(f_3);
    return 0;
}