#include<cstdio>
#include<hip/hip_runtime.h>
#include<cstring>
#include<cmath>

#define N 2048
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define TILE_SIZE 32
// 随机初始化size*size的矩阵
__host__ void init(int* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            matrix[i * size + j] = rand() % 10;
        }
    }
}

// 打印size行size列的矩阵
__host__ void printMatrix(int* matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            printf("%d ", matrix[i * size + j]);
        }
        printf("\n");
    }
}

// 使用共享内存进行矩阵转置
__global__ void transpose_shared(int* original, int* transposed, int size) {
    __shared__ int smem[TILE_SIZE][TILE_SIZE + 1];//避免bank冲突，设置TILE_SIZE+1列
    int bx = blockIdx.x * TILE_SIZE, by = blockIdx.y * TILE_SIZE;
    int tx = threadIdx.x, ty = threadIdx.y;
    int x = bx + tx, y = by + ty;

    if (x < size && y < size) {
        smem[ty][tx] = original[y * size + x];
    }
    __syncthreads();
    if (bx + ty < size && by + tx < size) {
        transposed[(bx + ty) * size + (by + tx)] = smem[tx][ty];
    }
}

int main(){
    int *original_matrix, *transposed_matrix;
    hipHostMalloc((void**)&original_matrix, sizeof(int) * N * N, hipHostMallocDefault); // 使用 cudaMallocHost 分配原矩阵内存
    hipHostMalloc((void**)&transposed_matrix, sizeof(int) * N * N, hipHostMallocDefault); // 使用 cudaMallocHost 分配转置后的矩阵内存
    init(original_matrix, N);   // 初始化原矩阵
    
    printf("原矩阵:\n");
    printMatrix(original_matrix,N);
    dim3 dimBlock(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
    dim3 dimGrid((N+BLOCK_DIM_X-1)/BLOCK_DIM_X, (N+BLOCK_DIM_Y-1)/BLOCK_DIM_Y, 1);
    
    //创建事件
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    hipEventRecord(start, 0); // 记录事件
    transpose_shared<<<dimGrid, dimBlock>>>(original_matrix, transposed_matrix, N);
    hipEventRecord(end, 0); // 记录事件
    hipEventSynchronize(end); // 同步
    float shared_time = 0;
    hipEventElapsedTime(&shared_time, start, end); // 计时
    
    
    printf("使用共享内存进行转置后的矩阵:\n");
    printMatrix(transposed_matrix,N);
    printf("线程块大小:(%d,%d) 矩阵规模:%d*%d 访存方式:共享内存 using time:%f ms\n", BLOCK_DIM_X,BLOCK_DIM_Y,N,N,shared_time);
    
    // 释放内存
    hipHostFree(original_matrix);
    hipHostFree(transposed_matrix);
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    return 0;
    
    
}